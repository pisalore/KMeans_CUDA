#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "vector_functions.hpp"
#include "hip/hip_vector_types.h"
#include "hip/hip_vector_types.h"
#include "hip/device_functions.h" 
#include "commonDefines.h"
#include <device_atomic_functions.h>

__device__ float euclideianDistance(float x1, float y1, float x2, float y2){
    float sum = float(pow(x1 - x2, 2) + pow(y1 - y2, 2));
    float distance = float(sqrt(sum));
    return distance;
}

__global__ void cuda_kMeans_clearAll(float* inputSums_x, float* inputSums_y, int* inputCounts) {
	int tx = threadIdx.x;
	inputSums_x[tx] = 0;
	inputSums_y[tx] = 0;
	inputCounts[tx] = 0;
}

__global__ void cuda_kMeans_CalculateDistances(float* points, float* inputClusters_x, float* inputClusters_y, int* clustersCount, float* outputSums_x, float* outputSums_y, int inputDimension, int vectorDimension, int clusterDimension) {
	int tx = threadIdx.x;
	int row = blockIdx.x * blockDim.x + tx;
	int it = row * vectorDimension;
	float distance;
	float minDistance;
	int clusterIndex;

	float2 point;
	float2 cluster;

	if (row < inputDimension) {
		point = make_float2(points[it], points[it + 1]); //load input point
		minDistance = 10000;
		clusterIndex = 0;

		for (int j = 0; j < clusterDimension; j++) {
			cluster = make_float2(inputClusters_x[j], inputClusters_y[j]); //from central gpu memory
			distance = euclideianDistance(point.x, point.y, cluster.x, cluster.y);

			if (distance < minDistance) {
				minDistance = distance;
				clusterIndex = j;
			}

		}
	
		atomicAdd(&outputSums_x[clusterIndex], point.x);
		atomicAdd(&outputSums_y[clusterIndex], point.y);
		atomicAdd(&clustersCount[clusterIndex], 1);
		
	}

}

__global__ void cuda_kMeans_updateCentroids(float* inputClusters_x, float* inputClusters_y, float* inputSums_x, float* inputSums_y, int* inputCounts) {
	int cluster = threadIdx.x;
	int count = max(1, inputCounts[cluster]);
	inputClusters_x[cluster] = inputSums_x[cluster] / count;
	inputClusters_y[cluster] = inputSums_y[cluster] / count;
}


extern "C"
void cuda_kMeans_CalculateDistances_wrapper(float* points, float* inputClusters_x, float* inputClusters_y, int* clustersCount, float* outputSums_x, float* outputSums_y, int inputDimension, int vectorDimension, int clusterDimension, dim3 gridDim, dim3 blockDim) {
	cuda_kMeans_CalculateDistances << <gridDim, blockDim >> > (points, inputClusters_x, inputClusters_y, clustersCount, outputSums_x, outputSums_y, inputDimension, vectorDimension, clusterDimension);
}

extern "C"
void cuda_kMeans_UpdateClusters_wrapper(float* inputClusters_x, float* inputClusters_y, float* inputSums_x, float* inputSums_y, int* inputCounts, dim3 gridDim, dim3 blockDim) {
	cuda_kMeans_updateCentroids << <gridDim, blockDim >> > (inputClusters_x, inputClusters_y, inputSums_x, inputSums_y, inputCounts);
}

extern "C"
void cuda_kMeans_ClearAll_wrapper(float* inputSums_x, float* inputSums_y, int* inputCounts, dim3 gridDim, dim3 blockDim) {
	cuda_kMeans_clearAll << <gridDim, blockDim >> > (inputSums_x, inputSums_y, inputCounts);
}



